#include "hip/hip_runtime.h"
/* Implements a threadsafe binary heap for use on a GPU
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>

// includes, project
#include "cutil.h"

#define BASETABLE_SIZE_MAX		10000000
#define MAX_ENTRIES		(2<<20)


__device__ __inline__ void lock(int *mutex ) {
	while( atomicCAS( mutex, 0, 1 ) != 0 );
}

__device__ __inline__ void unlock(int *mutex) {
	*mutex=0;
}

typedef int key_t;
typedef int value_t;

struct TableEntry {
	key_t mKey;
	value_t mValue;
	unsigned mNext;
};

typedef struct TableEntry tTableEntry;

struct BaseEntry {
       unsigned mIndex;
       int mLock;
};

struct HashTable {
	BaseEntry mValues[BASETABLE_SIZE_MAX];
};

typedef struct HashTable tHashTable;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, const char** argv);
__global__ void kernel_buildtable_atomic( tHashTable *__restrict__ g_hashtable, tTableEntry *__restrict__ g_entrypool, unsigned *__restrict__ g_keys, int g_baseTableSize );

extern "C"
int computeGold( int* gpuData, const int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, const char** argv)
{
    runTest( argc, argv);

    CUT_EXIT(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest( int argc, const char **argv)
{
    hipDeviceProp_t deviceProp;
    deviceProp.major = 0;
    deviceProp.minor = 0;
    int dev;
    CUT_DEVICE_INIT(argc, argv);
    CUDA_SAFE_CALL(hipChooseDevice(&dev, &deviceProp));
    CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev));

    if(deviceProp.major > 1 || deviceProp.minor > 0)
    {
        printf("Using Device %d: \"%s\"\n", dev, deviceProp.name);
        CUDA_SAFE_CALL(hipSetDevice(dev));
    }
    else
    {
        printf("There is no device supporting CUDA compute capability 1.1. Hopefully using emu\n");
        //CUT_EXIT(argc, argv);
    }


    unsigned int timer = 0;
    CUT_SAFE_CALL( cutCreateTimer( &timer));
    CUT_SAFE_CALL( cutStartTimer( timer));

    srand(2011);  // set seed for rand()

    int numThreads = 192;
    int numBlocks = 120;
    int baseTableSize = 8192;
    cutGetCmdLineArgumenti(argc, argv, "numThreads", &numThreads); 
    cutGetCmdLineArgumenti(argc, argv, "numBlocks", &numBlocks); 
    cutGetCmdLineArgumenti(argc, argv, "hashEntries", &baseTableSize);
    assert(numThreads >= 0); 
    assert(numBlocks >= 0); 
    assert(MAX_ENTRIES > (numThreads * numBlocks + 1));
    assert(baseTableSize >= 0 && baseTableSize <= BASETABLE_SIZE_MAX);
    
    printf("Number of hash entries = %u\n", baseTableSize);
    printf("Number of threads = %u\n", numThreads*numBlocks);

    // allocate host copy:
    tHashTable* h_hashtable = (tHashTable*)(calloc(1, sizeof(tHashTable)));
    tTableEntry* h_entries = (tTableEntry*)(calloc(MAX_ENTRIES, sizeof(tTableEntry)));
    unsigned* h_keys = (unsigned*)(calloc(MAX_ENTRIES, sizeof(unsigned)));

    // and device copy
    tHashTable* d_hashtable;
    tTableEntry* d_entries;
    unsigned* d_keys;

    // Build keys
    for(unsigned i=0; i<MAX_ENTRIES; i++) {
        h_keys[i] = rand();
    }

    CUDA_SAFE_CALL( hipMalloc( (void**) &d_hashtable, sizeof(tHashTable)));
    CUDA_SAFE_CALL( hipMemcpy( d_hashtable, h_hashtable, sizeof(tHashTable), hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_entries, sizeof(tTableEntry)*MAX_ENTRIES ) );    
    CUDA_SAFE_CALL( hipMemcpy( d_entries, h_entries, sizeof(tTableEntry)*MAX_ENTRIES, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_keys, sizeof(unsigned)*MAX_ENTRIES ) );
    CUDA_SAFE_CALL( hipMemcpy( d_keys, h_keys, sizeof(unsigned)*MAX_ENTRIES, hipMemcpyHostToDevice) );
    
    // execute the first kernel, this throws some data into the kernel for testing...
    kernel_buildtable_atomic<<<numBlocks, numThreads>>>(d_hashtable, d_entries, d_keys, baseTableSize);

    CUT_CHECK_ERROR("Kernel execution failed");
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    //Copy result from device to host
    CUDA_SAFE_CALL( hipMemcpy( h_hashtable, d_hashtable, sizeof(tHashTable),
    hipMemcpyDeviceToHost) );
    CUDA_SAFE_CALL( hipMemcpy( h_entries, d_entries,
    sizeof(tTableEntry)*MAX_ENTRIES, hipMemcpyDeviceToHost) );

    #ifdef DEBUG
    for( int i = 0; i < MAX_ENTRIES; ++i ) {
        if( h_entries[i].mValue ) 
            printf(" %u : %u ->  %u\n", i, h_entries[i].mKey, h_entries[i].mValue );
    }
    #endif 

    #define DEBUG
    // error checking 
    int nInsertedEntries = 0; 
    for (int h = 0; h < baseTableSize; h++) {
        unsigned entry_id = h_hashtable->mValues[h].mIndex; 
        while (entry_id != 0) {
            tTableEntry& tentry = h_entries[entry_id]; 
            #ifdef DEBUG
            if ( (tentry.mKey != h_keys[tentry.mValue]) || (tentry.mKey % baseTableSize != h)) {
               printf(" table[%d] -> %u : %u -> %u\n", h, entry_id, tentry.mKey, tentry.mValue); 
            }
            #else
            assert(tentry.mKey == h_keys[tentry.mValue]); // key-value consistency
            assert((tentry.mKey % baseTableSize) == h); // key-hash consistency
            #endif
            entry_id = tentry.mNext; 
            nInsertedEntries += 1;
        }
    }
    printf("nInsertedEntries = %d\n", nInsertedEntries); 
    assert(nInsertedEntries == (numThreads * numBlocks)); 

    CUT_SAFE_CALL( cutStopTimer( timer));
    printf( "Processing time: %f (ms)\n", cutGetTimerValue( timer));
    CUT_SAFE_CALL( cutDeleteTimer( timer));
   
    // cleanup memory
    free(h_hashtable);
    CUDA_SAFE_CALL(hipFree(d_hashtable));

    printf("TEST PASSED\n"); 
}


__device__ __forceinline__ void add_to_hash_atomic( tHashTable * g_hashtable, tTableEntry *  g_entrypool, unsigned key, unsigned value, int g_baseTableSize )
{
    const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned hash = key % g_baseTableSize;
    unsigned pool_slot = tid+1; // reserve zero for null
 
   BaseEntry *base = &g_hashtable->mValues[hash];
   tTableEntry *ent = &g_entrypool[pool_slot]; 
   
   // need something fancier if each thread can have more than one entry
   // TODO: implement something like Hoard (ASPLOS 2000) for CUDA
   while( atomicCAS( &base->mLock, 0, 1 ) != 0 );
   ent->mKey = key;
   ent->mValue = value;
   ent->mNext = base->mIndex;
   g_hashtable->mValues[hash].mIndex = pool_slot;
   base->mLock=0;
}


__global__ void kernel_buildtable_atomic( tHashTable *__restrict__ g_hashtable, tTableEntry *__restrict__ g_entrypool, unsigned *__restrict__ g_keys, int g_baseTableSize ) 
{
    const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned key, value;

   key = g_keys[tid+1];
   value = tid+1;
   add_to_hash_atomic(g_hashtable,g_entrypool,key,value,g_baseTableSize); 
}
