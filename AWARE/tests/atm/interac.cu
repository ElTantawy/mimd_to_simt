#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "cutil.h"


// #define DEBUG
// Only one of these should be enabled at a time. MKaech
//#define TM_SYNC
#define ATOMIC_SYNC
//#define NO_SYNC

#define NUM_ACCOUNTS 1000000
#define NUM_TRANSACTIONS 122880

#define THREADS_PER_BLOCK_X 192
#define THREADS_PER_BLOCK_Y 1
#define THREADS_PER_BLOCK_Z 1

#define BLOCKS_PER_GRID_X	120
#define BLOCKS_PER_GRID_Y	1
#define BLOCKS_PER_GRID_Z	1	// As of CUDA 2.0 this dimension MUST be 1.  MKaech

#define THREADS_PER_BLOCK	(THREADS_PER_BLOCK_X * THREADS_PER_BLOCK_Y * THREADS_PER_BLOCK_Z)
#define TOTAL_THREADS		(THREADS_PER_BLOCK * BLOCKS_PER_GRID_X * BLOCKS_PER_GRID_Y * BLOCKS_PER_GRID_Z)

// these macros are for use in the shader!
#define BLOCK_ID			( blockIdx.x + (BLOCKS_PER_GRID_X * blockIdx.y) + (BLOCKS_PER_GRID_X * BLOCKS_PER_GRID_Y * blockIdx.z) )
#define THREAD_ID			( (THREADS_PER_BLOCK * BLOCK_ID) + threadIdx.x + (THREADS_PER_BLOCK_X * threadIdx.y) + (THREADS_PER_BLOCK_X * THREADS_PER_BLOCK_Y * threadIdx.z) )


__device__ __inline__ void lock(int *mutex ) { 
        while( atomicCAS( mutex, 0, 1 ) != 0 );
}

__device__ __inline__ void unlock(int *mutex) {
        *mutex=0;
}

__device__ __inline__ bool trylock(int *mutex){
	if(atomicCAS( mutex, 0, 1 ) == 0 ){
		return true;
	}
	return false;
}


struct account
{
	int lock;
	int balance;
};

struct transaction
{
	int amount;
	int src_account;
	int dest_account;
};


__global__ void interac_atomic( account* __restrict__ accounts, transaction *__restrict__ transactions, int numTransactions)
{
	int id = THREAD_ID; 
	for(int index = id; index < numTransactions; index += TOTAL_THREADS)
	{
		transaction* action = &transactions[index];
		account* src = &accounts[action->src_account];
		account* dest = &accounts[action->dest_account];
		
		// sanity check
		if(action->src_account == action->dest_account)
		{
			continue;
		}
	
		// acquire locks
      		account* lock1;
      		account* lock2; 
      		if (src > dest) {
         		lock1 = src; 
         		lock2 = dest;
      		} else {
         		lock2 = src; 
         		lock1 = dest;
      		}
	
		int transaction_done = 0;
		while(!transaction_done) {
		        while( atomicCAS( &lock1->lock, 0, 1 ) != 0 );                	
	        	if(atomicCAS( &lock2->lock, 0, 1 ) == 0 ){
               			src->balance -= action->amount;
               			dest->balance += action->amount;
               			lock2->lock=0;
			        lock1->lock=0;		
        		       transaction_done = 1;
            		} else {
			        lock1->lock=0;		
			}
	      }	
	}
}

void interac_gold(account* __restrict__  accounts, transaction* __restrict__ transactions, int num_transactions)
{
	for(int i = 0; i < num_transactions; ++i)
	{
		transaction* action = &transactions[i];
		account* src = &accounts[action->src_account];
		account* dest = &accounts[action->dest_account];
		
		src->balance -= action->amount;
		dest->balance += action->amount;
	}
}
	
int main(int argc, const char** argv)
{
    printf("Initializing...\n");
    CUT_DEVICE_INIT(argc, argv);

    bool useTM = false;
    useTM = cutCheckCmdLineFlag(argc, argv, "tm"); 
    srand(2009);  // set seed for rand()

    // allocate host memory for accounts
    unsigned int accounts_size = sizeof(account) * NUM_ACCOUNTS;
	unsigned int transactions_size = sizeof(transaction) * NUM_TRANSACTIONS;
    account* host_accounts = (account*)malloc(accounts_size);
	account* gold_accounts = (account*)malloc(accounts_size);
	transaction* host_transactions = (transaction*)malloc(transactions_size);

	// create random account balances
    for (int i = 0; i < NUM_ACCOUNTS; ++i)
	{
		host_accounts[i].lock = 0;
        host_accounts[i].balance = (int) fmod((float)rand(),100.0f);
		
		gold_accounts[i].lock = 0;
		gold_accounts[i].balance = host_accounts[i].balance;
#ifdef DEBUG
		printf( "acct%u : $%d\n", i, host_accounts[i].balance );
#endif
	}
	
	// create random transaction pairs
	for (int i = 0; i < NUM_TRANSACTIONS; ++i)
	{
		host_transactions[i].amount = (int) fmod((float)rand(),50.0f);
		host_transactions[i].src_account = rand() % NUM_ACCOUNTS;	
		host_transactions[i].dest_account = rand() % NUM_ACCOUNTS;
#ifdef DEBUG
		printf( "%u : $%d from acct%u => to acct%u\n", 
			i, host_transactions[i].amount, 
			host_transactions[i].src_account, 
			host_transactions[i].dest_account );
#endif		
		// make sure src != dest
		while(host_transactions[i].src_account == host_transactions[i].dest_account)
		{
			host_transactions[i].dest_account = rand() % NUM_ACCOUNTS;
		}
	}

    // allocate device memory
    account* device_accounts;
	transaction* device_transactions;
    CUDA_SAFE_CALL(hipMalloc((void**) &device_accounts, accounts_size));
    CUDA_SAFE_CALL(hipMalloc((void**) &device_transactions, transactions_size));

    // copy host memory to device
    CUDA_SAFE_CALL(hipMemcpy(device_accounts, host_accounts, accounts_size, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL(hipMemcpy(device_transactions, host_transactions, transactions_size, hipMemcpyHostToDevice) );
    
    // setup execution parameters
	dim3 block_size(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y, THREADS_PER_BLOCK_Z);
	dim3 grid_size(BLOCKS_PER_GRID_X, BLOCKS_PER_GRID_Y, BLOCKS_PER_GRID_Z);
	
	printf("Beginning kernel execution...\n");
	
    // create and start timer
    unsigned int timer = 0;
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    CUT_SAFE_CALL(cutStartTimer(timer));

    // execute the kernel
    interac_atomic<<< grid_size, block_size >>>(device_accounts, device_transactions, NUM_TRANSACTIONS);
	
    hipDeviceSynchronize();

    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");

    // copy result from device to host
    CUDA_SAFE_CALL(hipMemcpy(host_accounts, device_accounts, accounts_size, hipMemcpyDeviceToHost) );
	
    // stop and destroy timer
    CUT_SAFE_CALL(cutStopTimer(timer));
    printf("Kernel processing time: %f (ms) \n", cutGetTimerValue(timer));
    CUT_SAFE_CALL(cutDeleteTimer(timer));
	
	printf("Computing gold results...\n");
	
    unsigned int timer_cpu = 0;
    CUT_SAFE_CALL(cutCreateTimer(&timer_cpu));
    CUT_SAFE_CALL(cutStartTimer(timer_cpu));
    interac_gold(gold_accounts, host_transactions, NUM_TRANSACTIONS);
    CUT_SAFE_CALL(cutStopTimer(timer_cpu));
    printf("Gold result processing time: %f (ms) \n", cutGetTimerValue(timer_cpu));
    CUT_SAFE_CALL(cutDeleteTimer(timer_cpu));
	
	printf("Comparing results...\n");

    // check result
	bool success = true;
    for (int i = 0; i < NUM_ACCOUNTS; ++i)
	{
		if(gold_accounts[i].balance != host_accounts[i].balance)
		{
			success = false;
			printf("Difference found in account %d: Gold = %d, Kernel = %d\n", i, gold_accounts[i].balance, host_accounts[i].balance);
		}
	}
	
	printf("Test %s\n", (success ? "PASSED! All account balances were correct." : "FAILED!"));

    // clean up memory
    free(host_accounts);
	free(gold_accounts);
	free(host_transactions);
    CUDA_SAFE_CALL(hipFree(device_accounts));
	CUDA_SAFE_CALL(hipFree(device_transactions));
	
    CUT_EXIT(argc, argv);
}
