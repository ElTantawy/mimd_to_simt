#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include "cutil.h"
#define WARP_SIZE 32

__global__ void div_bar(int* a, int * mutex)
{
    unsigned tid = blockIdx.x *blockDim.x + threadIdx.x;
    if(tid%2){
	a[(tid+1)%WARP_SIZE]++;
	__syncthreads();
	a[tid]+=3;
    }else{
	a[(tid+1)%WARP_SIZE]--;
	__syncthreads();
	a[tid]-=3;
    }
}

int main()
{
    // allocate host copy:
    int* a = (int*)(calloc(WARP_SIZE, sizeof(int)));
    int* mutex = (int*)(calloc(1,sizeof(int)));

    // and device copy
    int* d_a;
    int *d_mutex;
    
    // Build keys
    for(unsigned i=0; i<WARP_SIZE; i++) {
        a[i] = 1;
    }

    CUDA_SAFE_CALL( hipMalloc( (void**) &d_a, sizeof(int)*WARP_SIZE));
    CUDA_SAFE_CALL( hipMemcpy( d_a, a, sizeof(int)*WARP_SIZE, hipMemcpyHostToDevice) );

    CUDA_SAFE_CALL( hipMalloc( (void**) &d_mutex, sizeof(int)));
    CUDA_SAFE_CALL( hipMemcpy( d_mutex, mutex, sizeof(int), hipMemcpyHostToDevice) );

    div_bar<<<1, WARP_SIZE>>>(d_a, d_mutex);
    
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    //Copy result from device to host
    CUDA_SAFE_CALL( hipMemcpy(a ,d_a, sizeof(int)*WARP_SIZE,hipMemcpyDeviceToHost) );

    for( unsigned int i = 0; i < (WARP_SIZE/10); ++i)
    {   
        for(unsigned int j=0; j < 10; j++)
                printf("%d ",a[i*10+j]);
        printf("\n");
    }
	
}
